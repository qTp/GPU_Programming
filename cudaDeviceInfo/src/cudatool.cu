#include "../header/cudatool.h"

void cudaErr(hipError_t err){
    if (err != hipSuccess){
      printf( "Cuda error in file '%s' in line %i : %s.",
      __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
		}
}

#include "hip/hip_runtime.h"
#include "../header/cudatool.h"

//ErrorHandling for CUDA functions
hipError_t cudaErrT(hipError_t err, int line, char* file){
 //, int line, char* file){
#if defined(DEBUG) || defined(_DEBUG)
    if (err != hipSuccess){
      printf( "\n*** Cuda error in file '%s' in line %i : %s. ***\n\n", file, line, hipGetErrorString(err));
      exit(EXIT_FAILURE);
		}
#endif
    return err;
}

//Matrix gegen einander testen
void compareMatrix(double *P1, double *P2, int N, char name1[25], char name2[25]){
#if defined(DEBUG)||defined(_DEBUG)
	double epsilon = 1.0e-8; //Fehlertoleranz
	// int match = 1;
	for (int i = 0; i < N; ++i){
		if (abs(P1[i] - P2[i]) > epsilon){
			// match = 0;
			printf("Arrays do not match!\nCompare between %s & %s!\n", name1, name2);
			printf("host:%5.10f gpu:%5.10f at Element %d\n\n", P1[i], P2[i], i);
			break;
		}
	}
	//if (match) printf("Arrays match! Compare between %s & %s.\n\n", name1, name2);
#endif
}

//Matrix gegen einander testen
void compareMatrix(float *P1, float *P2, int N, char name1[25], char name2[25]){
#if defined(DEBUG)||defined(_DEBUG)
	double epsilon = 1.0e-8; //Fehlertoleranz
	// int match = 1;
	for (int i = 0; i < N; ++i){
		if (abs(P1[i] - P2[i]) > epsilon){
			// match = 0;
			printf("Arrays do not match!\nCompare between %s & %s!\n", name1, name2);
			printf("host:%5.10f gpu:%5.10f at Element %d\n\n", P1[i], P2[i], i);
			break;
		}
	}
	// if (match) printf("Arrays match! Compare between %s & %s.\n\n", name1, name2);
#endif
}

//Matrix fuellen
void initMatrix(float *ip, int size){
	//random seed erstellen
	time_t t;
	srand((unsigned)time(&t));
	//Matrix auffuellen
	for (int i = 0; i < size; ++i){
		ip[i] = (float)(rand() & 0xFF) / 100.0f;
	}
}

//Ausgabe welcher Teil gestartet wird
void preProcess(char _name[40]){
  printf("\n*** Starte %s ***\n", _name );
}
//Ausgabe der Ergebnisse
void postProcess(int sumReps, int size, double tElapsed){
    printf("Time elapsed: %.5f ms\n", 1e3*(tElapsed / sumReps ));
    printf("Bandwidth: %.5f GB/s\n", (( (2 * sizeof(float) * size * sumReps) / (1024 * 1024 * 1024)) / tElapsed ) );
}

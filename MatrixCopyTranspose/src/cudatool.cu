#include "../header/cudatool.h"

hipError_t cudaErrT(hipError_t err, int line, char* file){
 //, int line, char* file){
#if defined(DEBUG) || defined(_DEBUG)
    if (err != hipSuccess){
      printf( "\n*** Cuda error in file '%s' in line %i : %s. ***\n\n", file, line, hipGetErrorString(err));
      assert(err != hipSuccess);
		}
#endif
    return err;
}

#include "../header/main.h"

int main(int argc, char**argv){
  int matrixWidth = 0;
  int matrixHeight = 0;
  int nReps = 0;
  dim3 blockDim(0);
  dim3 gridDim(0);
  float *inputMatrix; //inputMatrix
  float *ctrTMatrix; // compare and proof MatrixTranspose
  float *ctrCMatrix; // compare and proof MatrixCopy

  //DevicesProperties auslesen
  int deviceCount = 0;
  hipDeviceProp_t *dev_prop;
  cudaErr(hipGetDeviceCount(&deviceCount));
  dev_prop = (hipDeviceProp_t*) malloc (deviceCount*sizeof(hipDeviceProp_t));
  cudaErr(hipGetDeviceProperties(dev_prop,0));

  //ende

  if(argc == 4){ // 0:_Name; 1:width; 2:height; 3:NumberOfRepetition;
    matrixWidth = atoi(argv[1]);
    matrixHeight = atoi(argv[2]);
    nReps = atoi(argv[3]);

    blockDim.x = TILE_DIM;
    blockDim.y = BLOCK_ROWS;
    blockDim.z = 1;

    gridDim.x = matrixWidth/(TILE_DIM);
    gridDim.y = matrixWidth/(TILE_DIM);
    gridDim.z = 1;

    printf("\nGPU Device: %s\n", dev_prop->name );
    printf("WarpSize: %d, ThreadsPerMP: %d\n", dev_prop->warpSize, dev_prop->maxThreadsPerMultiProcessor);
    printf("\nMatrix dimension: width=%d, height=%d\n", matrixWidth, matrixHeight);
    printf("Wiederholungen fuer Zeitmessung: %d\n", nReps);
    printf("Grid dimension: x=%d, y=%d, z=%d\n",gridDim.x, gridDim.y, gridDim.z);
    printf("Block dimension: x=%d, y=%d, z=%d\n",blockDim.x, blockDim.y, blockDim.z);

  }else{
    printf("Wrong paramter!\n\n");
    exit(EXIT_FAILURE);
  }

  //speicherbedarf bestimmen
  int sizeMatrix = matrixWidth * matrixHeight;
  int memSizeMatrix = ( sizeof(float) * sizeMatrix );

  //Speicher holen
  inputMatrix = (float*)malloc(memSizeMatrix);
  ctrTMatrix = (float*)malloc(memSizeMatrix);
  ctrCMatrix = (float*)malloc(memSizeMatrix);

  //create inputMatrix
  initMatrix(inputMatrix, sizeMatrix);
  //create control Matrix for COPY test!
  measureAndBuildserialCopy(ctrCMatrix, inputMatrix, sizeMatrix, nReps);
  //create control Matrix for TRANSPOSE test!
  measureAndBuildserialTranspose(ctrTMatrix, inputMatrix, sizeMatrix, nReps);
  //create and measure CopyKernel
  measureCopyKernel(inputMatrix, ctrCMatrix, gridDim, blockDim, matrixWidth, matrixHeight, nReps);

  //Aufraeumen nicht vergessen!!!
  free(ctrTMatrix);
  free(ctrCMatrix);
  free(inputMatrix);
  free(dev_prop);
  //GPU Device zuruecksetzten
  hipDeviceReset();
  //OS mitteilen das wir durch sind.
  return EXIT_SUCCESS;
}

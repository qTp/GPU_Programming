#include "hip/hip_runtime.h"
#include "../header/main.h"

int main(int argc, char**argv){
  int matrixWidth = 0;
  int matrixHeight = 0;
  int nReps = 0;
  dim3 blockDim(0);
  dim3 gridDim(0);
  float *inputMatrix; //inputMatrix
  float *ctrTMatrix; // compare and proof MatrixTranspose
  float *ctrCMatrix; // compare and proof MatrixCopy
  float *goldMatrix; // je nach Kernel Func, entweder ctrTMatrix oder ctrCmatrix
  //Kernel function pointer
  _kernel_ kernelFunc;
  //name of the kernel function
  char *funcName;

  if(argc == 4){ // 0:_Name; 1:width; 2:height; 3:NumberOfRepetition;
    matrixWidth = atoi(argv[1]);
    matrixHeight = atoi(argv[2]);
    nReps = atoi(argv[3]);

    blockDim.x = TILE_DIM;
    blockDim.y = BLOCK_ROWS;
    blockDim.z = 1;

    gridDim.x = matrixWidth/(TILE_DIM);
    gridDim.y = matrixWidth/(TILE_DIM);
    gridDim.z = 1;
  }else{
    printf("Wrong paramter!\n\n");
    exit(EXIT_FAILURE);
  }
  //speicherbedarf bestimmen
  int sizeMatrix = matrixWidth * matrixHeight;
  int memSizeMatrix = ( sizeof(float) * sizeMatrix );
  //DevicesProperties auslesen
  int deviceCount = 0;
  size_t freeMem, totalMem;
  hipDeviceProp_t *dev_prop;
  cudaErr(hipGetDeviceCount(&deviceCount));
  dev_prop = (hipDeviceProp_t*) malloc (deviceCount*sizeof(hipDeviceProp_t));
  cudaErr(hipGetDeviceProperties(dev_prop,0));
  cudaErr(hipMemGetInfo(&freeMem, &totalMem));
  //ende
  
  printf("\nGPU Device: %s\n", dev_prop->name );
  float gMemGB = (dev_prop->totalGlobalMem / BYTE_TO_GBYTE );
  printf("WarpSize: %d, ThreadsPerMP: %d globalMem: %.2f GB\n",
          dev_prop->warpSize,
          dev_prop->maxThreadsPerMultiProcessor,
          gMemGB );
  printf("Free = %Iu, Total = %Iu\n", freeMem, totalMem);
  /*
  In this calculation, we convert the memory clock rate to Hz,
  multiply it by the interface width (divided by 8, to convert bits to bytes)
  and multiply by 2 due to the double data rate.
   Finally, we divide by 1e9 to convert the result to GB/s.
   http://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
  */
  printf("theoretical bandwidth: %.2f GB/s\n", dev_prop->memoryClockRate * 1e3 * (dev_prop->memoryBusWidth / 8) * 2  / 1e9);
  printf("Wiederholungen fuer Zeitmessung: %d\n", nReps);
  printf("Matrix dimension: width=%d, height=%d, requiredMem: %d\n", matrixWidth, matrixHeight, 2*memSizeMatrix);
  printf("Grid(%d,%d,%d)",gridDim.x, gridDim.y, gridDim.z);
  printf(" Block(%d,%d,%d)\n\n",blockDim.x, blockDim.y, blockDim.z);

  //Speicher holen
  inputMatrix = (float*)malloc(memSizeMatrix);
  ctrTMatrix = (float*)malloc(memSizeMatrix);
  ctrCMatrix = (float*)malloc(memSizeMatrix);

  //create inputMatrix
  initMatrix(inputMatrix, sizeMatrix);

#if defined(SERIAL)
  //create control Matrix for COPY test!
  measureAndBuildserialCopy(ctrCMatrix, inputMatrix, matrixWidth, matrixHeight, nReps);
  //create control Matrix for TRANSPOSE test!
  measureAndBuildserialTranspose(ctrTMatrix, inputMatrix, matrixWidth, matrixHeight, nReps);
#else
  serialCopy(ctrCMatrix, inputMatrix, matrixWidth, matrixHeight, 1);
  serialTranspose(ctrTMatrix, inputMatrix, matrixWidth, matrixHeight, 1);
#endif

  //schleife ueber alle Kernel Funktionen!!!
  // i = anzahl an KernelFunktionen ;-)
  for(int i = 0; i<2; ++i){
    switch(i){
      case 0:
        //create and measure CopyKernel
        kernelFunc = &copyMatrix;
        funcName = "copyMatrix\0";
        goldMatrix = ctrCMatrix;
        break;
      case 1:
        kernelFunc = &transposeMatrix;
        funcName = "transposeMatrix\0";
        goldMatrix = ctrTMatrix;
        break;
    }

    measureKernelOMP(inputMatrix, goldMatrix, gridDim, blockDim, matrixWidth, matrixHeight, nReps,funcName, kernelFunc);

    measureKernel(inputMatrix, goldMatrix, gridDim, blockDim, matrixWidth, matrixHeight, nReps,funcName, kernelFunc);
  }
  //Aufraeumen nicht vergessen!!!
  free(inputMatrix);
  free(ctrTMatrix);
  free(ctrCMatrix);
  //free(goldMatrix);
  free(dev_prop);
  //GPU Device zuruecksetzten
  hipDeviceReset();
  //OS mitteilen das wir durch sind.
  return EXIT_SUCCESS;
}

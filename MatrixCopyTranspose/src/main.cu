#include "hip/hip_runtime.h"
#include "../header/main.h"

int main(int argc, char**argv){
  int matrixWidth = 0;
  int matrixHeight = 0;
  int nReps = 0;
  dim3 blockDim(0);
  dim3 gridDim(0);
  float *inputMatrix; //inputMatrix
  float *ctrTMatrix; // compare and proof MatrixTranspose
  float *ctrCMatrix; // compare and proof MatrixCopy
  float *goldMatrix; // je nach Kernel Func, entweder ctrTMatrix oder ctrCmatrix
  //Kernel function pointer
  _kernel_ kernelFunc;
  //name of the kernel function
  char *funcName;

  //DevicesProperties auslesen
  int deviceCount = 0;
  hipDeviceProp_t *dev_prop;
  cudaErr(hipGetDeviceCount(&deviceCount));
  dev_prop = (hipDeviceProp_t*) malloc (deviceCount*sizeof(hipDeviceProp_t));
  cudaErr(hipGetDeviceProperties(dev_prop,0));

  //ende

  if(argc == 4){ // 0:_Name; 1:width; 2:height; 3:NumberOfRepetition;
    matrixWidth = atoi(argv[1]);
    matrixHeight = atoi(argv[2]);
    nReps = atoi(argv[3]);

    blockDim.x = TILE_DIM;
    blockDim.y = BLOCK_ROWS;
    blockDim.z = 1;

    gridDim.x = matrixWidth/(TILE_DIM);
    gridDim.y = matrixWidth/(TILE_DIM);
    gridDim.z = 1;

    printf("\nGPU Device: %s\n", dev_prop->name );
    printf("WarpSize: %d, ThreadsPerMP: %d\n", dev_prop->warpSize, dev_prop->maxThreadsPerMultiProcessor);
    //TODO theoretical bandwidth bestimmen!!
    /*
    In this calculation, we convert the memory clock rate to Hz,
    multiply it by the interface width (divided by 8, to convert bits to bytes)
    and multiply by 2 due to the double data rate.
     Finally, we divide by 1e9 to convert the result to GB/s.

     http://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
    */
    printf("theoretical bandwidth: %.2f GB/s\n", dev_prop->memoryClockRate * 1e3 * (dev_prop->memoryBusWidth / 8) * 2  / 1e9);
    printf("Matrix dimension: width=%d, height=%d\n", matrixWidth, matrixHeight);
    printf("Wiederholungen fuer Zeitmessung: %d\n", nReps);
    printf("Grid dimension: x=%d, y=%d, z=%d\n",gridDim.x, gridDim.y, gridDim.z);
    printf("Block dimension: x=%d, y=%d, z=%d\n\n",blockDim.x, blockDim.y, blockDim.z);

  }else{
    printf("Wrong paramter!\n\n");
    exit(EXIT_FAILURE);
  }

  //speicherbedarf bestimmen
  int sizeMatrix = matrixWidth * matrixHeight;
  int memSizeMatrix = ( sizeof(float) * sizeMatrix );

  //Speicher holen
  inputMatrix = (float*)malloc(memSizeMatrix);
  ctrTMatrix = (float*)malloc(memSizeMatrix);
  ctrCMatrix = (float*)malloc(memSizeMatrix);

  //create inputMatrix
  initMatrix(inputMatrix, sizeMatrix);

#if defined(DEBUG)||defined(_DEBUG)
  //create control Matrix for COPY test!
  measureAndBuildserialCopy(ctrCMatrix, inputMatrix, sizeMatrix, nReps);
  //create control Matrix for TRANSPOSE test!
  measureAndBuildserialTranspose(ctrTMatrix, inputMatrix, sizeMatrix, nReps);
#else
  serialCopy(ctrCMatrix, inputMatrix, sizeMatrix, 1);
  serialTranspose(ctrTMatrix, inputMatrix, sizeMatrix, 1);
#endif

  //schleife ueber alle Kernel Funktionen!!!
  // i = anzahl an KernelFunktionen ;-)
  for(int i = 0; i<2; ++i){
    switch(i){
      case 0:
        //create and measure CopyKernel
        kernelFunc = &copyMatrix;
        funcName = "copyMatrix\0";
        goldMatrix = ctrCMatrix;
        break;
      case 1:
        kernelFunc = &transposeMatrix;
        funcName = "transposeMatrix\0";
        goldMatrix = ctrTMatrix;
        break;
    }

    measureKernelOMP(inputMatrix, goldMatrix, gridDim, blockDim, matrixWidth, matrixHeight, nReps,funcName, kernelFunc);

    measureKernel(inputMatrix, goldMatrix, gridDim, blockDim, matrixWidth, matrixHeight, nReps,funcName, kernelFunc);
  }
  //Aufraeumen nicht vergessen!!!
  free(inputMatrix);
  free(ctrTMatrix);
  free(ctrCMatrix);
  //free(goldMatrix);
  free(dev_prop);
  //GPU Device zuruecksetzten
  hipDeviceReset();
  //OS mitteilen das wir durch sind.
  return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"

#include <stdbool.h>

#include <math.h>
#include <time.h>
#include <omp.h>

#include "../header/cudatool.h"

#define BLOCK_WIDTH 16


//parallel MatrixMult
__global__ void cudaMatrixMult(float *d_M, float *d_N, double *d_P, int width){
	//Berechne die Reihe/Zeile (ROW)
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	//Berechne die Spalte (COLUMN)
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	//Nur berechnen wenn die Zeile/Spalte noch in der Matrix ist! Wichtig!
	//Da hier auch Threads erstellt werden die nicht in der Matrix liegen.
	if ((row < width) && (col < width)){
		double Pvalue = 0;
		//jeder Thread berechnet genau ein Element der Ergebnismatrix
		for (int k = 0; k < width; k++){
			Pvalue += d_M[row*width + k] * d_N[k*width + col];
		}
		d_P[row*width + col] = Pvalue;
	}
}

//serial MatrixMult
void serialMatrixMult(float *M, float *N, double *P, int width){
	for (int i = 0; i < width; i++){
		for (int j = 0; j < width; j++){
			double sum = 0;
			for (int k = 0; k < width; k++){
				sum +=  M[i*width + k] * N[k*width + j];
			}
			P[i*width + j] = sum;
		}
	}
}

//Matrix fuellen
void initMatrix(float *ip, int size){
	//random seed erstellen
	time_t t;
	srand((unsigned)time(&t));
	//Matrix auffuellen
	for (int i = 0; i < size; i++){
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

//Matrix gegen einander testen
bool checkMatrix(double *Pserial, double *Pkernel, int N){
	double epsilon = 1.0e-8; //Fehlertoleranz
	bool match = 1;
	for (int i = 0; i < N; i++){
		if (abs(Pserial[i] - Pkernel[i]) > epsilon){
			match = 0;
			printf("Arrays do not match!\n");
			printf("host:\n%5.10f\ngpu:\n%5.10f\nat Element %d\n", Pserial[i], Pkernel[i], i);
			break;
		}
	}
	if (match) printf("Arrays match.\n\n");
	return match;
}

int main(int argc, char **argv){
	int h_width;//Breite der Matrix
	int h_arraySize;// groesse der Matrix = width * width
	int memSize = 0;
	int memSizeErg = 0;

	//variable for time calc
	double tStart = 0;
	double tEnd = 0;

	float *h_M;//Matrix 1 -> host = cpu
	float *h_N;//Matrix 2 -> host = cpu
	float *d_M;//Matrix 1 -> device = gpu <- im Devices-Speicher!
	float *d_N;//Matrix 2 -> device = gpu <- im Devices-Speicher!
	double *h_Ps;//Ergebnis serial
	double *h_Pk;//Ergebnis kernel
	double *d_Pk;//Ergebnis kernel <- im Devices-Speicher!
	dim3 dimGrid, dimBlock;

	if(argc == 4){
		h_width = atoi(argv[1]);
		dimBlock.x = atoi(argv[2]);
		dimBlock.y = atoi(argv[3]);
		dimBlock.z = 1;
		//Aus Block Grid bestimmen.
		dimGrid.x = ceil((float) h_width / dimBlock.x);
		dimGrid.y = ceil((float) h_width / dimBlock.y);
		dimGrid.z = 1;

	}else{
		printf("Falsche Parameter Anzahl!\n width blockX blockY !");
		exit(-1);
	}

	//Bestimmen der KERNEL Parameter ( GRID Dim & BLOCK Dim )
//	numBlocks = ceil((float) h_width / BLOCK_WIDTH );
	printf("Blocks= %d|%d for w: %d , Grid: %d|%d\n", dimBlock.x, dimBlock.y, h_width, dimGrid.x,dimGrid.y);
	//dim3 dimGrid(numBlocks, numBlocks);
	//dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH);

	//mit der width die Array groesse berechen
	h_arraySize = h_width * h_width;

	//Speichergroesse bestimmen
	memSize = (sizeof(float)*h_arraySize);
	memSizeErg = sizeof(double)*h_arraySize;
	//Host Arrays allokieren
	h_M = (float*)malloc(memSize);
	h_N = (float*)malloc(memSize);

	//Host-Ergebnis Array initialisieren
	h_Ps = (double*)malloc(memSizeErg);
	h_Pk = (double*)malloc(memSizeErg);

	//devices array allokieren
	cudaErr(hipMalloc((void**)&d_M, memSize));
	cudaErr(hipMalloc((void**)&d_N, memSize));

	//Device-Ergebnis array initialisieren
	cudaErr(hipMalloc((void**)&d_Pk, memSizeErg));

	//Matrix mit zuf�lligen Werten fuellen
	initMatrix(h_M, h_arraySize);
	initMatrix(h_N, h_arraySize);

	//host array in Device-Array kopieren
	cudaErr(hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice));
	cudaErr(hipMemcpy(d_N, h_N, memSize, hipMemcpyHostToDevice));

	//Matrix zu Testzwecken serielle berchenen lassen & Zeitmessung
	//Zeitmessung implementieren!!!
	printf("Start CPU MatrixMult\n");
	tStart = omp_get_wtime();
	serialMatrixMult(h_M, h_N, h_Ps, h_width);
	tEnd = omp_get_wtime();
	printf("Finish CPU MatrixMult in %f ms\n\n",1.e3*(tEnd - tStart));

	//Matrix auf der GPU berechnen; am besten diverse GRID | BLOCK kompiationen Testen
	printf("Start GPU MatrixMult aufwaermen\n");
	tStart = omp_get_wtime();
	cudaMatrixMult<<<dimGrid, dimBlock>>>(d_M, d_N, d_Pk, h_width);
	cudaErr(hipDeviceSynchronize());
	tEnd = omp_get_wtime();
	printf("Finish GPU MatrixMult in %f ms\n\n", 1.e3*(tEnd - tStart));

	printf("Start GPU MatrixMult jetzt aber richtig XD\n");
	tStart = omp_get_wtime();
	cudaMatrixMult<<<dimGrid, dimBlock>>>(d_M, d_N, d_Pk, h_width);
	cudaErr(hipDeviceSynchronize());
	tEnd = omp_get_wtime();
	printf("Finish GPU MatrixMultin %f ms\n\n", 1.e3*(tEnd - tStart));

	//Ergebnis kopieren
	cudaErr(hipMemcpy(h_Pk, d_Pk, memSizeErg, hipMemcpyDeviceToHost));

	//Matrix testen
	checkMatrix(h_Ps, h_Pk, h_arraySize);

	//Alles befreien
	free(h_M);
	free(h_N);
	free(h_Ps);
	cudaErr(hipFree(d_M));
	cudaErr(hipFree(d_N));
	cudaErr(hipFree(d_Pk));

	//nicht vergessen ;-)
	hipDeviceReset();
	//Programm mit Erfolg beenden
	return 0;
}

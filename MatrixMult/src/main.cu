#include "hip/hip_runtime.h"
#include "../header/main.h"

//Matrix fuellen
void initMatrix(float *ip, int size){
	//random seed erstellen
	time_t t;
	srand((unsigned)time(&t));
	//Matrix auffuellen
	for (int i = 0; i < size; ++i){
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

//Matrix gegen einander testen
void checkMatrix(double *Pserial, double *Pkernel, int N, char string[40]){
	double epsilon = 1.0e-8; //Fehlertoleranz
	int match = 1;
	for (int i = 0; i < N; ++i){
		if (abs(Pserial[i] - Pkernel[i]) > epsilon){
			match = 0;
			printf("Arrays do not match between %s!\n", string);
			printf("host:\n%5.10f\ngpu:\n%5.10f\nat Element %d\n", Pserial[i], Pkernel[i], i);
			break;
		}
	}
	if (match) printf("Arrays match between %s.\n\n", string);
}

int main(int argc, char **argv){

	int h_width;//Breite der Matrix
	int h_arraySize;// groesse der Matrix = width * width
	int memSize = 0;
	int memSizeErg = 0;
	char cBetween[20];
	//variable for time calc
	double tStart = 0;
	double tEnd = 0;

	float *h_M;//Matrix 1 -> host = cpu
	float *h_N;//Matrix 2 -> host = cpu
	float *d_M;//Matrix 1 -> device = gpu <- im Devices-Speicher!
	float *d_N;//Matrix 2 -> device = gpu <- im Devices-Speicher!
	double *h_Ps;//Ergebnis serial
	double *h_Pk;//Ergebnis kernel without shared mem
	double *h_PkSmem;//Ergebnis kernel with shared mem
	double *d_Pk;//Ergebnis kernel <- im Devices-Speicher!
	dim3 dimGrid, dimBlock, dimGridSMEM, dimBlockSMEM;

	if(argc == 4){
		h_width = atoi(argv[1]);
		dimBlock.x = atoi(argv[2]);
		dimBlock.y = atoi(argv[3]);
		dimBlock.z = 1;
		//Aus Block Grid bestimmen.
		dimGrid.x = ceil((float) h_width / dimBlock.x);
		dimGrid.y = ceil((float) h_width / dimBlock.y);
		dimGrid.z = 1;

		dimBlockSMEM.x = TILE_WIDTH;
		dimBlockSMEM.y = TILE_WIDTH;
		dimBlockSMEM.z = 1;

		dimGridSMEM.x = ceil((float)h_width / TILE_WIDTH);
		dimGridSMEM.y = ceil((float)h_width / TILE_WIDTH);
		dimGridSMEM.z = 1;

	}else{
		printf("Falsche Parameter Anzahl!\nwidth blockX blockY!\n\n");
		exit(-1);
	}

	//Bestimmen der KERNEL Parameter ( GRID Dim & BLOCK Dim )
	printf("Normal: Grid: %d|%d|%d and Block: %d|%d|%d for width: %d.\n", dimGrid.x,dimGrid.y,dimGrid.z, dimBlock.x, dimBlock.y,dimBlock.z, h_width);
	printf("SMem: Grid: %d|%d|%d and Block: %d|%d|%d for width: %d.\n", dimGridSMEM.x,dimGridSMEM.y,dimGridSMEM.z, dimBlockSMEM.x, dimBlockSMEM.y,dimBlockSMEM.z, h_width);

	//mit der width die Array groesse berechen
	h_arraySize = h_width * h_width;

	//Speichergroesse bestimmen
	memSize = sizeof(float)*h_arraySize;
	memSizeErg = sizeof(double)*h_arraySize;

	//Host Arrays allokieren
	h_M = (float*)malloc(memSize);
	h_N = (float*)malloc(memSize);

	//Host-Ergebnis Array initialisieren
	h_Ps = (double*)malloc(memSizeErg);
	h_Pk = (double*)malloc(memSizeErg);
	h_PkSmem = (double*)malloc(memSizeErg);

	//devices array allokieren
	cudaErr(hipMalloc((void**)&d_M, memSize));
	cudaErr(hipMalloc((void**)&d_N, memSize));

	//Device-Ergebnis array initialisieren
	cudaErr(hipMalloc((void**)&d_Pk, memSizeErg));

	//Matrix mit zufaelligen Werten fuellen
	initMatrix(h_M, h_arraySize);
	initMatrix(h_N, h_arraySize);

	//host array in Device-Array kopieren
	cudaErr(hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice));
	cudaErr(hipMemcpy(d_N, h_N, memSize, hipMemcpyHostToDevice));

	//Matrix zu Testzwecken serielle berchenen lassen & Zeitmessung
	//Zeitmessung implementieren!!!
	printf("Start CPU MatrixMult\n");
	tStart = omp_get_wtime();
	serialMatrixMult(h_M, h_N, h_Ps, h_width);
	tEnd = omp_get_wtime();
	printf("Finish CPU MatrixMult in %f ms\n\n",1.e3*(tEnd - tStart));
/*
	//Matrix auf der GPU berechnen; am besten diverse GRID | BLOCK kompiationen Testen
	printf("Start GPU MatrixMult aufwaermen\n");
	tStart = omp_get_wtime();
	cudaMatrixMult<<<dimGrid, dimBlock>>>(d_M, d_N, d_Pk, h_width);
	cudaErr(hipDeviceSynchronize());
	tEnd = omp_get_wtime();
	printf("Finish GPU MatrixMult in %f ms\n\n", 1.e3*(tEnd - tStart));

	printf("Start GPU MatrixMult jetzt aber richtig.\n");
	tStart = omp_get_wtime();
	cudaMatrixMult<<<dimGrid, dimBlock>>>(d_M, d_N, d_Pk, h_width);
	cudaErr(hipDeviceSynchronize());
	tEnd = omp_get_wtime();
	printf("Finish GPU MatrixMult in %f ms\n\n", 1.e3*(tEnd - tStart));
	//Ergebnis kopieren
	cudaErr(hipMemcpy(h_Pk, d_Pk, memSizeErg, hipMemcpyDeviceToHost));
	//Matrix testen
	strcpy(cBetween, "CPU - GPU w\\o smem");
	checkMatrix(h_Ps, h_Pk, h_arraySize,cBetween);
*/
	printf("Start GPU MatrixMult SharedMem aufwaermen.\n");
	tStart = omp_get_wtime();
	cudaMatrixMultWithSMem<<<dimGridSMEM, dimBlockSMEM>>>(d_M, d_N, d_Pk, h_width);
	cudaErr(hipDeviceSynchronize());
	tEnd = omp_get_wtime();
	printf("Finish GPU MatrixMult SharedMem in %f ms\n\n", 1.e3*(tEnd - tStart));

	printf("Start GPU MatrixMult SharedMem jetzt aber richtig..\n");
	tStart = omp_get_wtime();
	cudaMatrixMultWithSMem<<<dimGridSMEM, dimBlockSMEM>>>(d_M, d_N, d_Pk, h_width);
	cudaErr(hipDeviceSynchronize());
	tEnd = omp_get_wtime();
	printf("Finish GPU MatrixMult SharedMem in %f ms\n\n", 1.e3*(tEnd - tStart));
	//Ergebnis kopieren
	cudaErr(hipMemcpy(h_PkSmem, d_Pk, memSizeErg, hipMemcpyDeviceToHost));
	//Matrix testen
	strcpy(cBetween, "CPU - GPU with smem");
	checkMatrix(h_Ps, h_PkSmem, h_arraySize, cBetween);


	//Alles befreien
	free(h_M);
	free(h_N);
	free(h_Ps);
	cudaErr(hipFree(d_M));
	cudaErr(hipFree(d_N));
	cudaErr(hipFree(d_Pk));

	//nicht vergessen ;-)
	hipDeviceReset();
	//Programm mit Erfolg beenden
	return 0;
}

#include "../header/cudatool.h"

void cudaErrT(hipError_t err, int line, char* file){
    if (err != hipSuccess){
      printf( "\n*** Cuda error in file '%s' in line %i : %s. ***\n\n",
      file, line, hipGetErrorString(err));
      exit(EXIT_FAILURE);
		}
}
